#include "hip/hip_runtime.h"
#include "return_map.h"


//==========================================FILE=OPERATIONS====================================================




void debug_plot_points_2D(char f_name[], int Nx, int Ny, double *vals_x, double *vals_y){
    FILE *stream;
    stream=fopen(f_name,"w");

    for (int j = 0; j < Nx; ++j){
        for (int k = 0; k < Ny; ++k){

            fprintf(stream, "%le %le\n", vals_x[I2(j,k)], vals_y[I2(j,k)]);

        }
        
    }
    fclose(stream);

}


void debug_plot_points_3D(char f_name[], int Nx, int Ny, int Nz, double *vals_x, double *vals_y, double *vals_z){
    FILE *stream;
    stream=fopen(f_name,"w");

    for (int j = 0; j < Nx; ++j){
        for (int k = 0; k < Ny; ++k){
            for (int l = 0; l < Nz; ++l){

                fprintf(stream, "%le %le %le\n", vals_x[I3(j,k,l)], vals_y[I3(j,k,l)], vals_z[I3(j,k,l)]);
            }
        }
        
    }
    fclose(stream);
}

void debug_plot_points(char f_name[], int size, double *vals_x, double *vals_y, double *vals_z){
    FILE *stream;
    stream=fopen(f_name,"w");

    for (int j = 0; j < size; ++j){
        fprintf(stream, "%le %le %le\n", vals_x[j], vals_y[j], vals_z[j]);
    
    }
    fclose(stream);
}

void debug_plot_vector(char f_name[], double x0, double y0, double z0, double dx1, double dy1, double dz1, double scale){
    FILE *stream;
    stream=fopen(f_name,"w");

    double vec_x=dx1;
    double vec_y=dy1;
    double vec_z=dz1;

    normalize_vector(&vec_x, &vec_y, &vec_z);


    fprintf(stream, "%le %le %le\n", x0, y0, z0);
    fprintf(stream, "%le %le %le\n", x0+scale*vec_x, y0+scale*vec_y, z0+scale*vec_z);

    fclose(stream);
}



void debug_plot_vectors(char f_name[], int size, double *xp, double *yp, double *zp, double *vals_x, double *vals_y, double *vals_z, double scale){
    FILE *stream;
    stream=fopen(f_name,"w");

    for (int j = 0; j < size; ++j){
        fprintf(stream, "%le %le %le %le %le %le\n", xp[j], yp[j], zp[j], scale*vals_x[j], scale*vals_y[j], scale*vals_z[j]);
    
    }
    fclose(stream);
}


//==========================================FILE=OPERATIONS====================================================


inline __device__ construct_physical_vector(int Nx, int Ny, int Nz, int j_fixed, int k_fixed, int l_fixed, real x1, real x2, real x3, real *ux_d, real *uy_d, real *uz_d)
{
    ux_d[IN(j_fixed,k_fixed,l_fixed)]=x1;
    uy_d[IN(j_fixed,k_fixed,l_fixed)]=x2;
    uz_d[IN(j_fixed,k_fixed,l_fixed)]=x3;
}



void construct_plane_rectangular(int local_Nx, int local_Ny, real *local_x, real *local_y, real *local_z, real eps){

    int Nx=local_Nx;
    int Ny=local_Ny;
    real dx=2.0*eps/Nx;
    real dy=2.0*eps/Ny;

    for (int j = 0; j < Nx; ++j){
        real x=j*dx-eps;
        for (int k = 0; k < Ny; ++k){
            real y=k*dy-eps;    

            local_x[I2(j,k)]=x;
            local_y[I2(j,k)]=y;
            local_z[I2(j,k)]=0.0;
        }
    
    }

}



void create_matrix3_direct(real *Matrix, real cos_alpha, real sin_alpha, real ux, real uy, real uz){
    int Nx=3;
    
    double a=1.0/(1.0+cos_alpha);

/*
a=1.0/(1.0+cos_a)
N=np.array( [ [1 - a*(u[1]**2 + u[2]**2), a*u[0]*u[1] - u[2], u[1] + a*u[0]*u[2] ],
            [u[2] + a*u[0]*u[1], 1. - a*(u[0]**2 + u[2]**2), a*u[1]*u[2] - u[0]],
            [a*u[0]*u[2] - u[1],         u[0] + a*u[1]*u[2], 1. - a*(u[0]**2 + u[1]**2)]] )
*/


    Matrix[I2(0,0)]=1.0-a*(uy*uy+uz*uz);
    Matrix[I2(0,1)]=a*ux*uy-uz; 
    Matrix[I2(0,2)]=uy+a*ux*uz;
    
    Matrix[I2(1,0)]=uz+a*ux*uy; 
    Matrix[I2(1,1)]=1.0-a*(ux*ux+uz*uz); 
    Matrix[I2(1,2)]=a*uy*uz-ux;
    
    Matrix[I2(2,0)]=a*ux*uz-uy; 
    Matrix[I2(2,1)]=ux+a*uy*uz; 
    Matrix[I2(2,2)]=1.0-a*(ux*ux+uy*uy);


}

void MatrixVector_3_3(real *Matrix, real in_v1, real in_v2, real in_v3, real *out_v1, real *out_v2, real *out_v3){
    int Nx=3;
    
    out_v1[0]=Matrix[I2(0,0)]*in_v1+Matrix[I2(0,1)]*in_v2+Matrix[I2(0,2)]*in_v3;
    out_v2[0]=Matrix[I2(1,0)]*in_v1+Matrix[I2(1,1)]*in_v2+Matrix[I2(1,2)]*in_v3;
    out_v3[0]=Matrix[I2(2,0)]*in_v1+Matrix[I2(2,1)]*in_v2+Matrix[I2(2,2)]*in_v3;

}

real vector3_norm(real ux, real uy, real uz){

    return sqrt(ux*ux+uy*uy+uz*uz);
}


void vector3_normalize(real *ux, real *uy, real *uz){

    real norm=vector3_norm(ux[0], uy[0], uz[0]);

    if(norm==0.0) 
        norm=1.0;

    ux[0]/=norm;
    uy[0]/=norm;
    uz[0]/=norm;

}




void vector3_cross_product(real nx, real ny, real nz, real mx, real my, real mz, real *ux, real *uy, real *uz){

    ux[0]=ny*mz-nz*my;
    uy[0]=-(nx*mz-nz*mx);
    uz[0]=nx*my-ny*mx;

    //normalize_vector(ux, uy, uz);

}

real vector3_dot_product(real nx, real ny, real nz, real mx, real my, real mz){

    return nx*mx+ny*my+nz*mz;

}





void rotate(real *Matrix, real *plane_x, real *plane_y, real *plane_z, real *p_x, real *p_y, real *p_z, int size){


    for (int j = 0; j < size; ++j){
        real phase_x, phase_y, phase_z;
        MatrixVector_3_3(Matrix, plane_x[j], plane_y[j], plane_z[j], &phase_x, &phase_y, &phase_z);
        p_x[j]=phase_x;
        p_y[j]=phase_y;
        p_z[j]=phase_z;
    }


}


void translate_plane(real x0, real y0, real z0, int size, real *p_x, real *p_y, real *p_z){

    for(int j=0;j<size;++j){
        p_x[j]+=x0;
        p_y[j]+=y0;
        p_z[j]+=z0;
    }

}



void rotate_plane(real rhs_x, real rhs_y, real rhs_z, real plane_nx, real plane_ny, real plane_nz, real *Matrix, int size, real *plane_x, real *plane_y, real *plane_z, real *p_x, real *p_y, real *p_z){

    real nx=rhs_x, ny=rhs_y, nz=rhs_z; //vector of the RHS
    real mx=plane_nx, my=plane_ny, mz=plane_nz;
    //mx,my,mz are vectors of the normal to the translated 2D plane

    vector3_normalize(&nx, &ny, &nz);
    vector3_normalize(&mx, &my, &mz);
    
    real ux, uy, uz; //rotating axis vector
    vector3_cross_product(mx, my, mz, nx, ny, nz, &ux, &uy, &uz);
    real cos_alpha=vector3_dot_product(nx, ny, nz, mx, my, mz);
    real sin_alpha=vector3_norm(ux, uy, uz);

    create_matrix3_direct(Matrix, cos_alpha, sin_alpha, ux, uy, uz);
    //print_Matrix_3_3(Matrix);
    rotate(Matrix, plane_x, plane_y, plane_z, p_x, p_y, p_z, size);

}



void execute_return_map()
{








}