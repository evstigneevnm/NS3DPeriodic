#include "hip/hip_runtime.h"
#include "return_map.h"


//==========================================FILE=OPERATIONS====================================================

void debug_plot_points_2D(char f_name[], int Nx, int Ny, double *vals_x, double *vals_y){
    FILE *stream;
    stream=fopen(f_name,"w");

    for (int j = 0; j < Nx; ++j){
        for (int k = 0; k < Ny; ++k){

            fprintf(stream, "%le %le\n", vals_x[I2(j,k)], vals_y[I2(j,k)]);

        }
        
    }
    fclose(stream);

}


void debug_plot_points_3D(char f_name[], int Nx, int Ny, int Nz, double *vals_x, double *vals_y, double *vals_z){
    FILE *stream;
    stream=fopen(f_name,"w");

    for (int j = 0; j < Nx; ++j){
        for (int k = 0; k < Ny; ++k){
            for (int l = 0; l < Nz; ++l){

                fprintf(stream, "%le %le %le\n", vals_x[I3(j,k,l)], vals_y[I3(j,k,l)], vals_z[I3(j,k,l)]);
            }
        }
        
    }
    fclose(stream);
}

void debug_plot_points(char f_name[], int size, double *vals_x, double *vals_y, double *vals_z){
    FILE *stream;
    stream=fopen(f_name,"w");

    for (int j = 0; j < size; ++j){
        fprintf(stream, "%le %le %le\n", vals_x[j], vals_y[j], vals_z[j]);
    
    }
    fclose(stream);
}

void debug_plot_vector(char f_name[], double x0, double y0, double z0, double dx1, double dy1, double dz1, double scale){
    FILE *stream;
    stream=fopen(f_name,"w");

    double vec_x=dx1;
    double vec_y=dy1;
    double vec_z=dz1;

    normalize_vector(&vec_x, &vec_y, &vec_z);


    fprintf(stream, "%le %le %le\n", x0, y0, z0);
    fprintf(stream, "%le %le %le\n", x0+scale*vec_x, y0+scale*vec_y, z0+scale*vec_z);

    fclose(stream);
}



void debug_plot_vectors(char f_name[], int size, double *xp, double *yp, double *zp, double *vals_x, double *vals_y, double *vals_z, double scale){
    FILE *stream;
    stream=fopen(f_name,"w");

    for (int j = 0; j < size; ++j){
        fprintf(stream, "%le %le %le %le %le %le\n", xp[j], yp[j], zp[j], scale*vals_x[j], scale*vals_y[j], scale*vals_z[j]);
    
    }
    fclose(stream);
}


//==========================================FILE=OPERATIONS====================================================


inline __device__ construct_physical_vector_device(int Nx, int Ny, int Nz, int j_fixed, int k_fixed, int l_fixed, real x1, real x2, real x3, real *ux_d, real *uy_d, real *uz_d)
{
    ux_d[IN(j_fixed,k_fixed,l_fixed)]=x1;
    uy_d[IN(j_fixed,k_fixed,l_fixed)]=x2;
    uz_d[IN(j_fixed,k_fixed,l_fixed)]=x3;
}


void return_physical_vector3(int Nx, int Ny, int Nz, real *ux, real *uy, real *uz, real *v3x, real *v3y, real *v3z, int j_fixed, int k_fixed, int l_fixed){

    v3x[0]=ux[IN(j_fixed,k_fixed,l_fixed)];
    v3y[0]=uy[IN(j_fixed,k_fixed,l_fixed)];
    v3z[0]=uz[IN(j_fixed,k_fixed,l_fixed)];


}



void construct_plane_rectangular(int local_Nx, int local_Ny, real *local_x, real *local_y, real *local_z, real eps){

    int Nx=local_Nx;
    int Ny=local_Ny;
    real dx=2.0*eps/Nx;
    real dy=2.0*eps/Ny;

    for (int j = 0; j < Nx; ++j){
        real x=j*dx-eps;
        for (int k = 0; k < Ny; ++k){
            real y=k*dy-eps;    

            local_x[I2(j,k)]=x;
            local_y[I2(j,k)]=y;
            local_z[I2(j,k)]=0.0;
        }
    
    }

}



void create_matrix3_direct(real *Matrix, real cos_alpha, real sin_alpha, real ux, real uy, real uz){
    int Nx=3;
    
    double a=1.0/(1.0+cos_alpha);

/*
a=1.0/(1.0+cos_a)
N=np.array( [ [1 - a*(u[1]**2 + u[2]**2), a*u[0]*u[1] - u[2], u[1] + a*u[0]*u[2] ],
            [u[2] + a*u[0]*u[1], 1. - a*(u[0]**2 + u[2]**2), a*u[1]*u[2] - u[0]],
            [a*u[0]*u[2] - u[1],         u[0] + a*u[1]*u[2], 1. - a*(u[0]**2 + u[1]**2)]] )
*/


    Matrix[I2(0,0)]=1.0-a*(uy*uy+uz*uz);
    Matrix[I2(0,1)]=a*ux*uy-uz; 
    Matrix[I2(0,2)]=uy+a*ux*uz;
    
    Matrix[I2(1,0)]=uz+a*ux*uy; 
    Matrix[I2(1,1)]=1.0-a*(ux*ux+uz*uz); 
    Matrix[I2(1,2)]=a*uy*uz-ux;
    
    Matrix[I2(2,0)]=a*ux*uz-uy; 
    Matrix[I2(2,1)]=ux+a*uy*uz; 
    Matrix[I2(2,2)]=1.0-a*(ux*ux+uy*uy);


}

void MatrixVector_3_3(real *Matrix, real in_v1, real in_v2, real in_v3, real *out_v1, real *out_v2, real *out_v3){
    int Nx=3;
    
    out_v1[0]=Matrix[I2(0,0)]*in_v1+Matrix[I2(0,1)]*in_v2+Matrix[I2(0,2)]*in_v3;
    out_v2[0]=Matrix[I2(1,0)]*in_v1+Matrix[I2(1,1)]*in_v2+Matrix[I2(1,2)]*in_v3;
    out_v3[0]=Matrix[I2(2,0)]*in_v1+Matrix[I2(2,1)]*in_v2+Matrix[I2(2,2)]*in_v3;

}

real vector3_norm(real ux, real uy, real uz){

    return sqrt(ux*ux+uy*uy+uz*uz);
}


void vector3_normalize(real *ux, real *uy, real *uz){

    real norm=vector3_norm(ux[0], uy[0], uz[0]);

    if(norm==0.0) 
        norm=1.0;

    ux[0]/=norm;
    uy[0]/=norm;
    uz[0]/=norm;

}




void vector3_cross_product(real nx, real ny, real nz, real mx, real my, real mz, real *ux, real *uy, real *uz){

    ux[0]=ny*mz-nz*my;
    uy[0]=-(nx*mz-nz*mx);
    uz[0]=nx*my-ny*mx;

    //normalize_vector(ux, uy, uz);

}

real vector3_dot_product(real nx, real ny, real nz, real mx, real my, real mz){

    return nx*mx+ny*my+nz*mz;

}





void rotate(real *Matrix, real *plane_x, real *plane_y, real *plane_z, real *p_x, real *p_y, real *p_z, int size){


    for (int j = 0; j < size; ++j){
        real phase_x, phase_y, phase_z;
        MatrixVector_3_3(Matrix, plane_x[j], plane_y[j], plane_z[j], &phase_x, &phase_y, &phase_z);
        p_x[j]=phase_x;
        p_y[j]=phase_y;
        p_z[j]=phase_z;
    }


}


void translate_plane(real x0, real y0, real z0, int size, real *p_x, real *p_y, real *p_z){

    for(int j=0;j<size;++j){
        p_x[j]+=x0;
        p_y[j]+=y0;
        p_z[j]+=z0;
    }

}



void rotate_plane(real rhs_x, real rhs_y, real rhs_z, real plane_nx, real plane_ny, real plane_nz, real *Matrix, int size, real *plane_x, real *plane_y, real *plane_z, real *p_x, real *p_y, real *p_z){

    real nx=rhs_x, ny=rhs_y, nz=rhs_z; //vector of the RHS
    real mx=plane_nx, my=plane_ny, mz=plane_nz;
    //mx,my,mz are vectors of the normal to the translated 2D plane

    vector3_normalize(&nx, &ny, &nz);
    vector3_normalize(&mx, &my, &mz);
    
    real ux, uy, uz; //rotating axis vector
    vector3_cross_product(mx, my, mz, nx, ny, nz, &ux, &uy, &uz);
    real cos_alpha=vector3_dot_product(nx, ny, nz, mx, my, mz);
    real sin_alpha=vector3_norm(ux, uy, uz);

    create_matrix3_direct(Matrix, cos_alpha, sin_alpha, ux, uy, uz);
    //print_Matrix_3_3(Matrix);
    rotate(Matrix, plane_x, plane_y, plane_z, p_x, p_y, p_z, size);

}



void return_vector3_RHS(dim3 dimGrid, dim3 dimBlock, dim3 dimGrid_C, dim3 dimBlock_C, real dx, real dy, real dz, real Re, int Nx, int Ny, int Nz, int Mz, cudaComplex *ux_hat_d_plane, cudaComplex *uy_hat_d_plane, cudaComplex *uz_hat_d_plane,  cudaComplex *fx_hat_d, cudaComplex *fy_hat_d, cudaComplex *fz_hat_d, cudaComplex *Qx_hat_d, cudaComplex *Qy_hat_d, cudaComplex *Qz_hat_d, cudaComplex *div_hat_d, real* kx_nabla_d, real* ky_nabla_d, real *kz_nabla_d, real *din_diffusion_d, real *din_poisson_d, real *AM_11_d, real *AM_22_d, real *AM_33_d,  real *AM_12_d, real *AM_13_d, real *AM_23_d, cudaComplex *RHSx_hat_d, cudaComplex *RHSy_hat_d, cudaComplex *RHSz_hat_d, real *RHSx_d, real *RHSy_d, real *RHSz_d, int j_fixed, int k_fixed, int l_fixed, real *rhs_x, real *rhs_y, real *rhs_z){


    return_RHS(dimGrid, dimBlock, dimGrid_C, dimBlock_C,  dx,  dy,  dz,  Re,  Nx,  Ny,  Nz,  Mz, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d, kx_nabla_d,  ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d,  AM_12_d, AM_13_d, AM_23_d, RHSx_hat_d, RHSy_hat_d, RHSz_hat_d);

    iFFTN_Device(RHSx_hat_d, RHSx_d);
    iFFTN_Device(RHSy_hat_d, RHSy_d);
    iFFTN_Device(RHSz_hat_d, RHSz_d);

    host_device_real_cpy(RHSx, RHSx_d, Nx, Ny, Nz);
    host_device_real_cpy(RHSy, RHSy_d, Nx, Ny, Nz);
    host_device_real_cpy(RHSz, RHSz_d, Nx, Ny, Nz );

    return_physical_vector3(Nx, Ny, Nz, RHSx, RHSy, RHSz, rhs_x, rhs_y, rhs_z, j_fixed,  k_fixed, l_fixed);

}


void single_forward_step(dim3 dimGrid, dim3 dimBlock, dim3 dimGrid_C, dim3 dimBlock_C, real dx, real dy, real dz, real dt, real Re, int Nx, int Ny, int Nz, int Mz, cudaComplex *ux_hat_d_plane, cudaComplex *uy_hat_d_plane, cudaComplex *uz_hat_d_plane, cudaComplex *ux_hat_d_1, cudaComplex *uy_hat_d_1, cudaComplex *uz_hat_d_1,  cudaComplex *ux_hat_d_2, cudaComplex *uy_hat_d_2, cudaComplex *uz_hat_d_2,  cudaComplex *ux_hat_d_3, cudaComplex *uy_hat_d_3, cudaComplex *uz_hat_d_3,  cudaComplex *fx_hat_d, cudaComplex *fy_hat_d, cudaComplex *fz_hat_d, cudaComplex *Qx_hat_d, cudaComplex *Qy_hat_d, cudaComplex *Qz_hat_d, cudaComplex *div_hat_d, real* kx_nabla_d, real* ky_nabla_d, real *kz_nabla_d, real *din_diffusion_d, real *din_poisson_d, real *AM_11_d, real *AM_22_d, real *AM_33_d,  real *AM_12_d, real *AM_13_d, real *AM_23_d, real *ux_d_plane, real *uy_d_plane, real *uz_d_plane,  real *ux_plane, real *uy_plane, real *uz_plane, int j_fixed, int k_fixed, int l_fixed, real *point_x, real *point_y, real *point_z){


    RK3_SSP(dimGrid, dimBlock, dimGrid_C, dimBlock_C, dx, dy, dz, dt, Re, Nx, Ny, Nz, Mz, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane,  ux_hat_d_1, uy_hat_d_1, uz_hat_d_1,  ux_hat_d_2, uy_hat_d_2, uz_hat_d_2,  ux_hat_d_3, uy_hat_d_3, uz_hat_d_3,  fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d,  kx_nabla_d,  ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d,  AM_12_d, AM_13_d, AM_23_d);

    velocity_to_double(dimGrid, dimBlock, Nx, Ny, Nz, ux_hat_d_plane, ux_d_plane, uy_hat_d_plane, uy_d_plane, uz_hat_d_plane, uz_d_plane);
    
    host_device_real_cpy(ux_plane, ux_d_plane, Nx, Ny, Nz);
    host_device_real_cpy(uy_plane, uy_d_plane, Nx, Ny, Nz);
    host_device_real_cpy(uz_plane, uz_d_plane, Nx, Ny, Nz);

    return_physical_vector3(Nx, Ny, Nz, ux_plane, uy_plane, uz_plane, point_x, point_y, point_z, j_fixed,  k_fixed, l_fixed);

}

//0 select a point in the plane 
//1 call single_forward_step.
//2 If the condition of the interseciton is met, then we find the intersection point and store the result, else, goto 1.
//3 take next point in the plane
//4 goto 1.

void execute_return_map()
{








}