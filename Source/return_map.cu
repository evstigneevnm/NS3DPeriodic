#include "hip/hip_runtime.h"
#include "return_map.h"

void vector3_normalize(real *vec_x, real *vec_y, real *vec_z);

//==========================================FILE=OPERATIONS====================================================




void debug_plot_points_2D(char f_name[], int Nx, int Ny, double *vals_x, double *vals_y)
{
    FILE *stream;
    stream=fopen(f_name,"w");

    for (int j = 0; j < Nx; ++j){
        for (int k = 0; k < Ny; ++k){

            fprintf(stream, "%le %le\n", vals_x[I2(j,k)], vals_y[I2(j,k)]);

        }
        
    }
    fclose(stream);

}


void debug_plot_points_3D(char f_name[], int Nx, int Ny, int Nz, double *vals_x, double *vals_y, double *vals_z)
{
    FILE *stream;
    stream=fopen(f_name,"w");

    for (int j = 0; j < Nx; ++j){
        for (int k = 0; k < Ny; ++k){
            for (int l = 0; l < Nz; ++l){

                fprintf(stream, "%.16le %.16le %.16le\n", vals_x[I3(j,k,l)], vals_y[I3(j,k,l)], vals_z[I3(j,k,l)]);
            }
        }
        
    }
    fclose(stream);
}

void debug_plot_points(char f_name[], int size, double *vals_x, double *vals_y, double *vals_z)
{
    FILE *stream;
    stream=fopen(f_name,"w");

    for (int j = 0; j < size; ++j){
        fprintf(stream, "%.16le %.16le %.16le\n", vals_x[j], vals_y[j], vals_z[j]);
    
    }
    fclose(stream);
}


void plot_points_pos(char f_name[], int size, double *vals_x, double *vals_y, double *vals_z)
{
    FILE *stream;
    stream=fopen(f_name,"w");
    fprintf(stream, "View \"%s\"{\n", f_name);
    for (int j = 0; j < size; ++j){
        fprintf(stream, "SP(%.16le,%.16le,%.16le){%i};\n", vals_x[j], vals_y[j], vals_z[j],j);
    }
    fprintf(stream, "};");
    fclose(stream);
}



void debug_plot_vector(char f_name[], double x0, double y0, double z0, double dx1, double dy1, double dz1, double scale)
{
    FILE *stream;
    stream=fopen(f_name,"w");

    double vec_x=dx1;
    double vec_y=dy1;
    double vec_z=dz1;

    vector3_normalize(&vec_x, &vec_y, &vec_z);


    fprintf(stream, "%.16le %.16le %.16le\n", x0, y0, z0);
    fprintf(stream, "%.16le %.16le %.16le\n", x0+scale*vec_x, y0+scale*vec_y, z0+scale*vec_z);

    fclose(stream);
}



void debug_plot_vectors(char f_name[], int size, double *xp, double *yp, double *zp, double *vals_x, double *vals_y, double *vals_z, double scale)
{
    FILE *stream;
    stream=fopen(f_name,"w");

    for (int j = 0; j < size; ++j){
        fprintf(stream, "%.16le %.16le %.16le %.16le %.16le %.16le\n", xp[j], yp[j], zp[j], scale*vals_x[j], scale*vals_y[j], scale*vals_z[j]);
    
    }
    fclose(stream);
}


//==========================================FILE=OPERATIONS====================================================




void return_physical_vector3(int Nx, int Ny, int Nz, real *ux, real *uy, real *uz, real *v3x, real *v3y, real *v3z, int j_fixed, int k_fixed, int l_fixed)
{

    v3x[0]=ux[IN(j_fixed,k_fixed,l_fixed)];
    v3y[0]=uy[IN(j_fixed,k_fixed,l_fixed)];
    v3z[0]=uz[IN(j_fixed,k_fixed,l_fixed)];


}




void construct_plane_rectangular(int local_Nx, int local_Ny, real *local_x, real *local_y, real *local_z, real eps)
{

    int Nx=local_Nx;
    int Ny=local_Ny;
    real dx=2.0*eps/Nx;
    real dy=2.0*eps/Ny;

    for (int j = 0; j < Nx; ++j){
        real x=j*dx-eps;
        for (int k = 0; k < Ny; ++k){
            real y=k*dy-eps;    

            local_x[I2(j,k)]=x;
            local_y[I2(j,k)]=y;
            local_z[I2(j,k)]=0.0;
        }
    
    }

}



void create_matrix3_direct(real *Matrix, real cos_alpha, real sin_alpha, real ux, real uy, real uz){
    int Nx=3;
    
    double a=1.0/(1.0+cos_alpha);

/*
a=1.0/(1.0+cos_a)
N=np.array( [ [1 - a*(u[1]**2 + u[2]**2), a*u[0]*u[1] - u[2], u[1] + a*u[0]*u[2] ],
            [u[2] + a*u[0]*u[1], 1. - a*(u[0]**2 + u[2]**2), a*u[1]*u[2] - u[0]],
            [a*u[0]*u[2] - u[1],         u[0] + a*u[1]*u[2], 1. - a*(u[0]**2 + u[1]**2)]] )
*/


    Matrix[I2(0,0)]=1.0-a*(uy*uy+uz*uz);
    Matrix[I2(0,1)]=a*ux*uy-uz; 
    Matrix[I2(0,2)]=uy+a*ux*uz;
    
    Matrix[I2(1,0)]=uz+a*ux*uy; 
    Matrix[I2(1,1)]=1.0-a*(ux*ux+uz*uz); 
    Matrix[I2(1,2)]=a*uy*uz-ux;
    
    Matrix[I2(2,0)]=a*ux*uz-uy; 
    Matrix[I2(2,1)]=ux+a*uy*uz; 
    Matrix[I2(2,2)]=1.0-a*(ux*ux+uy*uy);


}

void MatrixVector_3_3(real *Matrix, real in_v1, real in_v2, real in_v3, real *out_v1, real *out_v2, real *out_v3){
    int Nx=3;
    
    out_v1[0]=Matrix[I2(0,0)]*in_v1+Matrix[I2(0,1)]*in_v2+Matrix[I2(0,2)]*in_v3;
    out_v2[0]=Matrix[I2(1,0)]*in_v1+Matrix[I2(1,1)]*in_v2+Matrix[I2(1,2)]*in_v3;
    out_v3[0]=Matrix[I2(2,0)]*in_v1+Matrix[I2(2,1)]*in_v2+Matrix[I2(2,2)]*in_v3;

}

real vector3_norm(real ux, real uy, real uz){

    return sqrt(ux*ux+uy*uy+uz*uz);
}




void vector3_normalize(real *ux, real *uy, real *uz){

    real norm=vector3_norm(ux[0], uy[0], uz[0]);

    if(norm==0.0) 
        norm=1.0;

    ux[0]/=norm;
    uy[0]/=norm;
    uz[0]/=norm;

}




void vector3_cross_product(real nx, real ny, real nz, real mx, real my, real mz, real *ux, real *uy, real *uz){

    ux[0]=ny*mz-nz*my;
    uy[0]=-(nx*mz-nz*mx);
    uz[0]=nx*my-ny*mx;

    //normalize_vector(ux, uy, uz);

}

real vector3_dot_product(real nx, real ny, real nz, real mx, real my, real mz){

    return nx*mx+ny*my+nz*mz;

}





void rotate(real *Matrix, real *plane_x, real *plane_y, real *plane_z, real *p_x, real *p_y, real *p_z, int size){


    for (int j = 0; j < size; ++j){
        real phase_x, phase_y, phase_z;
        MatrixVector_3_3(Matrix, plane_x[j], plane_y[j], plane_z[j], &phase_x, &phase_y, &phase_z);
        p_x[j]=phase_x;
        p_y[j]=phase_y;
        p_z[j]=phase_z;
    }


}


void translate_plane(real x0, real y0, real z0, int size, real *p_x, real *p_y, real *p_z){

    for(int j=0;j<size;++j){
        p_x[j]+=x0;
        p_y[j]+=y0;
        p_z[j]+=z0;
    }

}

real test_plane_location(real nx, real ny, real nz, real x0, real y0, real z0, real x, real y, real z){

    real vx=(x-x0);
    real vy=(y-y0);
    real vz=(z-z0);

    return(vector3_dot_product(nx, ny, nz, vx, vy, vz));
}


void rotate_plane(real rhs_x, real rhs_y, real rhs_z, real plane_nx, real plane_ny, real plane_nz, real *Matrix, int size, real *plane_x, real *plane_y, real *plane_z, real *p_x, real *p_y, real *p_z){

    real nx=rhs_x, ny=rhs_y, nz=rhs_z; //vector of the RHS
    real mx=plane_nx, my=plane_ny, mz=plane_nz;
    //mx,my,mz are vectors of the normal to the translated 2D plane

    vector3_normalize(&nx, &ny, &nz);
    vector3_normalize(&mx, &my, &mz);
    
    real ux, uy, uz; //rotating axis vector
    vector3_cross_product(mx, my, mz, nx, ny, nz, &ux, &uy, &uz);
    real cos_alpha=vector3_dot_product(nx, ny, nz, mx, my, mz);
    real sin_alpha=vector3_norm(ux, uy, uz);

    create_matrix3_direct(Matrix, cos_alpha, sin_alpha, ux, uy, uz);
    //print_Matrix_3_3(Matrix);
    rotate(Matrix, plane_x, plane_y, plane_z, p_x, p_y, p_z, size);

}



void return_vector3_RHS(dim3 dimGrid, dim3 dimBlock, dim3 dimGrid_C, dim3 dimBlock_C, real dx, real dy, real dz, real Re, int Nx, int Ny, int Nz, int Mz, cudaComplex *ux_hat_d, cudaComplex *uy_hat_d, cudaComplex *uz_hat_d,  cudaComplex *fx_hat_d, cudaComplex *fy_hat_d, cudaComplex *fz_hat_d, cudaComplex *Qx_hat_d, cudaComplex *Qy_hat_d, cudaComplex *Qz_hat_d, cudaComplex *div_hat_d, real* kx_nabla_d, real* ky_nabla_d, real *kz_nabla_d, real *din_diffusion_d, real *din_poisson_d, real *AM_11_d, real *AM_22_d, real *AM_33_d,  real *AM_12_d, real *AM_13_d, real *AM_23_d, cudaComplex *RHSx_hat_d, cudaComplex *RHSy_hat_d, cudaComplex *RHSz_hat_d, real *RHSx_d, real *RHSy_d, real *RHSz_d, real *RHSx, real *RHSy, real *RHSz, int j_fixed, int k_fixed, int l_fixed, real *rhs_x, real *rhs_y, real *rhs_z){


    return_RHS(dimGrid, dimBlock, dimGrid_C, dimBlock_C,  dx,  dy,  dz,  Re,  Nx,  Ny,  Nz,  Mz, ux_hat_d, uy_hat_d, uz_hat_d, fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d, kx_nabla_d,  ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d,  AM_12_d, AM_13_d, AM_23_d, RHSx_hat_d, RHSy_hat_d, RHSz_hat_d);

    velocity_to_double(dimGrid, dimBlock, Nx, Ny, Nz, RHSx_hat_d, RHSx_d, RHSy_hat_d, RHSy_d, RHSz_hat_d, RHSz_d);

    host_device_real_cpy(RHSx, RHSx_d, Nx, Ny, Nz);
    host_device_real_cpy(RHSy, RHSy_d, Nx, Ny, Nz);
    host_device_real_cpy(RHSz, RHSz_d, Nx, Ny, Nz);

    return_physical_vector3(Nx, Ny, Nz, RHSx, RHSy, RHSz, rhs_x, rhs_y, rhs_z, j_fixed, k_fixed, l_fixed);

}

void return_vector3_solution(int j_fixed,  int k_fixed, int l_fixed, dim3 dimGrid, dim3 dimBlock, int Nx, int Ny, int Nz, cudaComplex *ux_hat_d, cudaComplex *uy_hat_d, cudaComplex *uz_hat_d, real *ux_d, real *uy_d, real *uz_d, real *ux, real *uy, real *uz, real *point_x, real *point_y, real *point_z)
{

    velocity_to_double(dimGrid, dimBlock, Nx, Ny, Nz, ux_hat_d, ux_d, uy_hat_d, uy_d, uz_hat_d, uz_d);

    host_device_real_cpy(ux, ux_d, Nx, Ny, Nz);
    host_device_real_cpy(uy, uy_d, Nx, Ny, Nz);
    host_device_real_cpy(uz, uz_d, Nx, Ny, Nz);

    return_physical_vector3(Nx, Ny, Nz, ux, uy, uz, point_x, point_y, point_z, j_fixed,  k_fixed, l_fixed);

}



void return_vector3_RHS_curl(dim3 dimGrid, dim3 dimBlock, dim3 dimGrid_C, dim3 dimBlock_C, real dx, real dy, real dz, real Re, int Nx, int Ny, int Nz, int Mz, cudaComplex *ux_hat_d, cudaComplex *uy_hat_d, cudaComplex *uz_hat_d,  cudaComplex *fx_hat_d, cudaComplex *fy_hat_d, cudaComplex *fz_hat_d, cudaComplex *Qx_hat_d, cudaComplex *Qy_hat_d, cudaComplex *Qz_hat_d, cudaComplex *div_hat_d, real* kx_nabla_d, real* ky_nabla_d, real *kz_nabla_d, real *din_diffusion_d, real *din_poisson_d, real *AM_11_d, real *AM_22_d, real *AM_33_d,  real *AM_12_d, real *AM_13_d, real *AM_23_d, cudaComplex *RHSx_hat_d, cudaComplex *RHSy_hat_d, cudaComplex *RHSz_hat_d, real *RHSx_d, real *RHSy_d, real *RHSz_d, real *rot_x_d, real *rot_y_d, real *rot_z_d, real *rot_x, real *rot_y, real *rot_z, int j_fixed, int k_fixed, int l_fixed, real *rhs_x, real *rhs_y, real *rhs_z){


    return_RHS(dimGrid, dimBlock, dimGrid_C, dimBlock_C,  dx,  dy,  dz,  Re,  Nx,  Ny,  Nz,  Mz, ux_hat_d, uy_hat_d, uz_hat_d, fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d, kx_nabla_d,  ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d,  AM_12_d, AM_13_d, AM_23_d, RHSx_hat_d, RHSy_hat_d, RHSz_hat_d);

    velocity_to_double(dimGrid, dimBlock, Nx, Ny, Nz, RHSx_hat_d, RHSx_d, RHSy_hat_d, RHSy_d, RHSz_hat_d, RHSz_d);
    get_curl(dimGrid, dimBlock, Nx, Ny, Nz, dx, dy, dz, RHSx_d, RHSy_d, RHSz_d, rot_x_d, rot_y_d, rot_z_d);

    host_device_real_cpy(rot_x, rot_x_d, Nx, Ny, Nz);
    host_device_real_cpy(rot_y, rot_y_d, Nx, Ny, Nz);
    host_device_real_cpy(rot_z, rot_z_d, Nx, Ny, Nz);

    return_physical_vector3(Nx, Ny, Nz, rot_x, rot_y, rot_z, rhs_x, rhs_y, rhs_z, j_fixed, k_fixed, l_fixed);




}


void return_vector3_solution_curl(int j_fixed,  int k_fixed, int l_fixed, dim3 dimGrid, dim3 dimBlock, int Nx, int Ny, int Nz, real dx, real dy, real dz, cudaComplex *ux_hat_d, cudaComplex *uy_hat_d, cudaComplex *uz_hat_d, real *ux_d, real *uy_d, real *uz_d, real *rot_x_d, real *rot_y_d, real *rot_z_d,  real *rot_x, real *rot_y, real *rot_z, real *point_x, real *point_y, real *point_z)
{

    velocity_to_double(dimGrid, dimBlock, Nx, Ny, Nz, ux_hat_d, ux_d, uy_hat_d, uy_d, uz_hat_d, uz_d);
    get_curl(dimGrid, dimBlock, Nx, Ny, Nz, dx, dy, dz, ux_d, uy_d, uz_d, rot_x_d, rot_y_d, rot_z_d);

    host_device_real_cpy(rot_x, rot_x_d, Nx, Ny, Nz);
    host_device_real_cpy(rot_y, rot_y_d, Nx, Ny, Nz);
    host_device_real_cpy(rot_z, rot_z_d, Nx, Ny, Nz);


    return_physical_vector3(Nx, Ny, Nz, rot_x, rot_y, rot_z, point_x, point_y, point_z, j_fixed,  k_fixed, l_fixed);

}







void single_forward_step(dim3 dimGrid, dim3 dimBlock, dim3 dimGrid_C, dim3 dimBlock_C, real dx, real dy, real dz, real dt, real Re, int Nx, int Ny, int Nz, int Mz, cudaComplex *ux_hat_d_plane, cudaComplex *uy_hat_d_plane, cudaComplex *uz_hat_d_plane, cudaComplex *ux_hat_d_1, cudaComplex *uy_hat_d_1, cudaComplex *uz_hat_d_1,  cudaComplex *ux_hat_d_2, cudaComplex *uy_hat_d_2, cudaComplex *uz_hat_d_2,  cudaComplex *ux_hat_d_3, cudaComplex *uy_hat_d_3, cudaComplex *uz_hat_d_3,  cudaComplex *fx_hat_d, cudaComplex *fy_hat_d, cudaComplex *fz_hat_d, cudaComplex *Qx_hat_d, cudaComplex *Qy_hat_d, cudaComplex *Qz_hat_d, cudaComplex *div_hat_d, real* kx_nabla_d, real* ky_nabla_d, real *kz_nabla_d, real *din_diffusion_d, real *din_poisson_d, real *AM_11_d, real *AM_22_d, real *AM_33_d,  real *AM_12_d, real *AM_13_d, real *AM_23_d, real *ux_d_plane, real *uy_d_plane, real *uz_d_plane,  real *ux_plane, real *uy_plane, real *uz_plane, int j_fixed, int k_fixed, int l_fixed, real *point_x, real *point_y, real *point_z){


    RK3_SSP(dimGrid, dimBlock, dimGrid_C, dimBlock_C, dx, dy, dz, dt, Re, Nx, Ny, Nz, Mz, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane,  ux_hat_d_1, uy_hat_d_1, uz_hat_d_1,  ux_hat_d_2, uy_hat_d_2, uz_hat_d_2,  ux_hat_d_3, uy_hat_d_3, uz_hat_d_3,  fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d,  kx_nabla_d,  ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d,  AM_12_d, AM_13_d, AM_23_d);

    velocity_to_double(dimGrid, dimBlock, Nx, Ny, Nz, ux_hat_d_plane, ux_d_plane, uy_hat_d_plane, uy_d_plane, uz_hat_d_plane, uz_d_plane);
    
    host_device_real_cpy(ux_plane, ux_d_plane, Nx, Ny, Nz);
    host_device_real_cpy(uy_plane, uy_d_plane, Nx, Ny, Nz);
    host_device_real_cpy(uz_plane, uz_d_plane, Nx, Ny, Nz);

    return_physical_vector3(Nx, Ny, Nz, ux_plane, uy_plane, uz_plane, point_x, point_y, point_z, j_fixed,  k_fixed, l_fixed);

}



void single_forward_step_curl(dim3 dimGrid, dim3 dimBlock, dim3 dimGrid_C, dim3 dimBlock_C, real dx, real dy, real dz, real dt, real Re, int Nx, int Ny, int Nz, int Mz, cudaComplex *ux_hat_d, cudaComplex *uy_hat_d, cudaComplex *uz_hat_d, cudaComplex *ux_hat_d_1, cudaComplex *uy_hat_d_1, cudaComplex *uz_hat_d_1,  cudaComplex *ux_hat_d_2, cudaComplex *uy_hat_d_2, cudaComplex *uz_hat_d_2,  cudaComplex *ux_hat_d_3, cudaComplex *uy_hat_d_3, cudaComplex *uz_hat_d_3,  cudaComplex *fx_hat_d, cudaComplex *fy_hat_d, cudaComplex *fz_hat_d, cudaComplex *Qx_hat_d, cudaComplex *Qy_hat_d, cudaComplex *Qz_hat_d, cudaComplex *div_hat_d, real* kx_nabla_d, real* ky_nabla_d, real *kz_nabla_d, real *din_diffusion_d, real *din_poisson_d, real *AM_11_d, real *AM_22_d, real *AM_33_d,  real *AM_12_d, real *AM_13_d, real *AM_23_d, real *ux_d, real *uy_d, real *uz_d, real *rot_x_d, real *rot_y_d, real *rot_z_d,  real *rot_x, real *rot_y, real *rot_z, int j_fixed, int k_fixed, int l_fixed, real *point_x, real *point_y, real *point_z){


    RK3_SSP(dimGrid, dimBlock, dimGrid_C, dimBlock_C, dx, dy, dz, dt, Re, Nx, Ny, Nz, Mz, ux_hat_d, uy_hat_d, uz_hat_d,  ux_hat_d_1, uy_hat_d_1, uz_hat_d_1,  ux_hat_d_2, uy_hat_d_2, uz_hat_d_2,  ux_hat_d_3, uy_hat_d_3, uz_hat_d_3,  fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d,  kx_nabla_d,  ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d,  AM_12_d, AM_13_d, AM_23_d);

    velocity_to_double(dimGrid, dimBlock, Nx, Ny, Nz, ux_hat_d, ux_d, uy_hat_d, uy_d, uz_hat_d, uz_d);
   
    get_curl(dimGrid, dimBlock, Nx, Ny, Nz, dx, dy, dz, ux_d, uy_d, uz_d, rot_x_d, rot_y_d, rot_z_d);



    host_device_real_cpy(rot_x, rot_x_d, Nx, Ny, Nz);
    host_device_real_cpy(rot_y, rot_y_d, Nx, Ny, Nz);
    host_device_real_cpy(rot_z, rot_z_d, Nx, Ny, Nz);

    return_physical_vector3(Nx, Ny, Nz, rot_x, rot_y, rot_z, point_x, point_y, point_z, j_fixed,  k_fixed, l_fixed);

}




__global__ void construct_physical_vector_device(int Nx, int Ny, int Nz, int j_fixed, int k_fixed, int l_fixed,  real x_point, real y_point, real z_point, real *ux_d, real *uy_d, real *uz_d)
{


    ux_d[IN(j_fixed, k_fixed, l_fixed)]=x_point;
    uy_d[IN(j_fixed, k_fixed, l_fixed)]=y_point;
    uz_d[IN(j_fixed, k_fixed, l_fixed)]=z_point;


}


void construct_physical_vector(dim3 dimGrid, dim3 dimBlock, int Nx, int Ny, int Nz, int j_fixed, int k_fixed, int l_fixed,  real x_point, real y_point, real z_point, cudaComplex *ux_hat_d, cudaComplex *uy_hat_d, cudaComplex *uz_hat_d, real *ux_d_plane, real *uy_d_plane, real *uz_d_plane, cudaComplex *ux_hat_d_plane, cudaComplex *uy_hat_d_plane, cudaComplex *uz_hat_d_plane)
{

    velocity_to_double(dimGrid, dimBlock, Nx, Ny, Nz, ux_hat_d, ux_d_plane, uy_hat_d, uy_d_plane, uz_hat_d, uz_d_plane);

    construct_physical_vector_device<<<dimGrid, dimBlock>>>(Nx, Ny, Nz, j_fixed, k_fixed, l_fixed,  x_point, y_point, z_point, ux_d_plane , uy_d_plane, uz_d_plane);

    //Image_to_Domain(dimGrid, dimBlock, Nx, Ny, Nz, ux_d_plane, ux_hat_d_plane, uy_d_plane, uy_hat_d_plane, uz_d_plane, uz_hat_d_plane);
    Domain_to_Image(dimGrid, dimBlock,  Nx,  Ny,  Nz, ux_hat_d_plane, ux_d_plane, uy_hat_d_plane, uy_d_plane, uz_hat_d_plane, uz_d_plane);


}
//0 select a point in the plane 
//1 call single_forward_step.
//2 If the condition of the interseciton is met, then we find the intersection point and store the result, else, goto 1.
//3 take next point in the plane
//4 goto 1.

bool find_intersection(int steps, real x_0, real y_0, real z_0,  real *x_next, real x_prev, real *y_next, real y_prev, real *z_next, real z_prev, real rhs_x, real rhs_y, real rhs_z, int j_fixed, int k_fixed, int l_fixed, dim3 dimGrid, dim3 dimBlock, dim3 dimGrid_C, dim3 dimBlock_C, real dx, real dy, real dz, real dt, real Re, int Nx, int Ny, int Nz, int Mz, cudaComplex *ux_hat_d_plane, cudaComplex *uy_hat_d_plane, cudaComplex *uz_hat_d_plane, cudaComplex *ux_hat_d_plane_back, cudaComplex *uy_hat_d_plane_back, cudaComplex *uz_hat_d_plane_back, cudaComplex *ux_hat_d_1, cudaComplex *uy_hat_d_1, cudaComplex *uz_hat_d_1,  cudaComplex *ux_hat_d_2, cudaComplex *uy_hat_d_2, cudaComplex *uz_hat_d_2,  cudaComplex *ux_hat_d_3, cudaComplex *uy_hat_d_3, cudaComplex *uz_hat_d_3,  cudaComplex *fx_hat_d, cudaComplex *fy_hat_d, cudaComplex *fz_hat_d, cudaComplex *Qx_hat_d, cudaComplex *Qy_hat_d, cudaComplex *Qz_hat_d, cudaComplex *div_hat_d, real* kx_nabla_d, real* ky_nabla_d, real *kz_nabla_d, real *din_diffusion_d, real *din_poisson_d, real *AM_11_d, real *AM_22_d, real *AM_33_d,  real *AM_12_d, real *AM_13_d, real *AM_23_d, real *ux_d_plane, real *uy_d_plane, real *uz_d_plane,  real *ux_plane, real *uy_plane, real *uz_plane)
{

    const real rho=5.0e-1;
    bool return_flag=false;


    //note - "ux_hat_d_plane_back, uy_hat_d_plane_back, uz_hat_d_plane_back" are having previous timestep stored!

    copy_arrays(dimGrid_C, dimBlock_C, Nx, Ny, Nz,  ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, ux_hat_d_plane_back, uy_hat_d_plane_back, uz_hat_d_plane_back);

    single_forward_step(dimGrid, dimBlock, dimGrid_C, dimBlock_C, dx, dy, dz, dt, Re, Nx, Ny, Nz, Mz, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, ux_hat_d_1, uy_hat_d_1, uz_hat_d_1,  ux_hat_d_2, uy_hat_d_2, uz_hat_d_2, ux_hat_d_3, uy_hat_d_3, uz_hat_d_3, fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d, kx_nabla_d, ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d, AM_12_d, AM_13_d, AM_23_d, ux_d_plane, uy_d_plane, uz_d_plane, ux_plane, uy_plane, uz_plane, j_fixed, k_fixed, l_fixed, x_next, y_next, z_next);

    real test_vec_x=x_next[0]-x_prev, test_vec_y=y_next[0]-y_prev, test_vec_z=z_next[0]-z_prev;

    //crosses plane
    real sign_1=test_plane_location(rhs_x, rhs_y, rhs_z, x_0, y_0, z_0, x_prev, y_prev, z_prev);
    real sign_2=test_plane_location(rhs_x, rhs_y, rhs_z, x_0, y_0, z_0, x_next[0], y_next[0], z_next[0]); 
    
    //in the same direction
    real sign_3=vector3_dot_product(rhs_x, rhs_y, rhs_z, test_vec_x, test_vec_y, test_vec_z); 
    
    real vec_x=x_prev-x_0;
    real vec_y=y_prev-y_0;
    real vec_z=z_prev-z_0;
     //in the ball_rho
    real vec_norm=vector3_norm(vec_x, vec_y, vec_z);


    if((steps>3)&&(vec_norm<rho)&&(sign_1*sign_2<0.0)&&(sign_3>0.0)){
        real dt1=dt;
        real xn1=x_prev, yn1=y_prev, zn1=z_prev;
        real err_s=test_plane_location(rhs_x, rhs_y, rhs_z, x_0, y_0, z_0, x_next[0], y_next[0], z_next[0]);
        real del_s=test_plane_location(rhs_x, rhs_y, rhs_z, x_0, y_0, z_0, x_prev, y_prev, z_prev);
        real err=std::fabs(err_s);
        real del=std::fabs(del_s);
        
        int iter=0;
        real a_val=0.0;
        real b_val=dt1;
        while((std::fabs(err)>1.0e-12)&&(iter<500)){                        
            iter++;
            
            real m_val=0.5*(b_val-a_val);
            //dt1=dt1*(del)/(del+err);
            

            //restore previous step!
            copy_arrays(dimGrid_C, dimBlock_C, Nx, Ny, Nz, ux_hat_d_plane_back, uy_hat_d_plane_back, uz_hat_d_plane_back,  ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane);
            single_forward_step(dimGrid, dimBlock, dimGrid_C, dimBlock_C,  dx, dy, dz, /*!*/m_val/*!*/, Re,  Nx,  Ny,  Nz,  Mz, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, ux_hat_d_1, uy_hat_d_1, uz_hat_d_1,  ux_hat_d_2, uy_hat_d_2, uz_hat_d_2,  ux_hat_d_3, uy_hat_d_3, uz_hat_d_3,  fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d, kx_nabla_d, ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d,  AM_12_d, AM_13_d, AM_23_d, ux_d_plane, uy_d_plane, uz_d_plane,  ux_plane, uy_plane, uz_plane, j_fixed, k_fixed, l_fixed, &xn1, &yn1, &zn1);

            err=test_plane_location(rhs_x, rhs_y, rhs_z, x_0, y_0, z_0, xn1, yn1, zn1);

            if(err>0.0){
                b_val=m_val;
            }
            else{
                a_val=m_val;
                //shift base point
                copy_arrays(dimGrid_C, dimBlock_C, Nx, Ny, Nz,  ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, ux_hat_d_plane_back, uy_hat_d_plane_back, uz_hat_d_plane_back);      
            }

        }
        printf("\{%le,%i\}", err, iter);
        x_next[0]=xn1; y_next[0]=yn1; z_next[0]=zn1;
        return_flag=true;
    }

    return return_flag;
}







void execute_return_map(int j_fixed, int k_fixed, int l_fixed, dim3 dimGrid, dim3 dimBlock, dim3 dimGrid_C, dim3 dimBlock_C, real dx, real dy, real dz, real dt, real Re, int Nx, int Ny, int Nz, int Mz, cudaComplex *ux_hat_d, cudaComplex *uy_hat_d, cudaComplex *uz_hat_d, cudaComplex *ux_hat_d_1, cudaComplex *uy_hat_d_1, cudaComplex *uz_hat_d_1,  cudaComplex *ux_hat_d_2, cudaComplex *uy_hat_d_2, cudaComplex *uz_hat_d_2,  cudaComplex *ux_hat_d_3, cudaComplex *uy_hat_d_3, cudaComplex *uz_hat_d_3,  cudaComplex *fx_hat_d, cudaComplex *fy_hat_d, cudaComplex *fz_hat_d, cudaComplex *Qx_hat_d, cudaComplex *Qy_hat_d, cudaComplex *Qz_hat_d, cudaComplex *div_hat_d, real* kx_nabla_d, real* ky_nabla_d, real *kz_nabla_d, real *din_diffusion_d, real *din_poisson_d, real *AM_11_d, real *AM_22_d, real *AM_33_d,  real *AM_12_d, real *AM_13_d, real *AM_23_d)
{

    int Nrad=7, Nphi=7;
    int number_of_points=Nrad*Nphi;
    real x_0, y_0, z_0;
    real rhs_x, rhs_y, rhs_z;

    real *x_loc, *y_loc, *z_loc, *p_x, *p_y, *p_z, *Matrix;
    real *v_x, *v_y, *v_z;
    real *vx_loc, *vy_loc;
    allocate_real(Nrad, Nphi, 1, 11, &x_loc, &y_loc, &z_loc, &p_x, &p_y, &p_z, &v_x, &v_y, &v_z, &vx_loc, &vy_loc);
    Matrix=allocate_d(3,3,1);
    
    real *ux, *uy, *uz;
    allocate_real(Nx, Ny, Nz, 3, &ux, &uy, &uz);

    cudaComplex *ux_hat_d_plane, *uy_hat_d_plane, *uz_hat_d_plane;
    cudaComplex *ux_hat_d_plane_back, *uy_hat_d_plane_back, *uz_hat_d_plane_back;
    cudaComplex *ux_hat_d_shift, *uy_hat_d_shift, *uz_hat_d_shift;
    real *ux_d_plane, *uy_d_plane, *uz_d_plane;


    device_allocate_all_complex(Nx, Ny, Mz, 3, &ux_hat_d_plane, &uy_hat_d_plane, &uz_hat_d_plane);
    device_allocate_all_complex(Nx, Ny, Mz, 3, &ux_hat_d_plane_back, &uy_hat_d_plane_back, &uz_hat_d_plane_back);
    device_allocate_all_complex(Nx, Ny, Mz, 3, &ux_hat_d_shift, &uy_hat_d_shift, &uz_hat_d_shift);
    device_allocate_all_real(Nx, Ny, Nz, 3, &ux_d_plane, &uy_d_plane, &uz_d_plane);


    //obtaining the RHS vector at a currect solution point
    return_vector3_RHS(dimGrid,  dimBlock,  dimGrid_C, dimBlock_C, dx, dy, dz, Re, Nx, Ny,  Nz, Mz, ux_hat_d, uy_hat_d, uz_hat_d,  fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d, kx_nabla_d,  ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d, AM_12_d, AM_13_d, AM_23_d, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, ux_d_plane, uy_d_plane, uz_d_plane, ux, uy, uz, j_fixed,  k_fixed, l_fixed, &rhs_x, &rhs_y, &rhs_z);

    //get a point of x0,y0,z0 from the solution
    return_vector3_solution(j_fixed,  k_fixed, l_fixed, dimGrid, dimBlock, Nx, Ny, Nz, ux_hat_d, uy_hat_d, uz_hat_d, ux_d_plane, uy_d_plane, uz_d_plane, ux, uy, uz, &x_0, &y_0, &z_0);


/* 
    =============================================
    ADVANCING FUTHER TO GET ANOTHER CUT PLANE!!!
    =============================================
*/ 
    copy_arrays(dimGrid_C, dimBlock_C, Nx, Ny, Nz, ux_hat_d, uy_hat_d, uz_hat_d, ux_hat_d_shift, uy_hat_d_shift, uz_hat_d_shift); 
    for(int t=0;t<930;t++){
        RK3_SSP(dimGrid, dimBlock, dimGrid_C, dimBlock_C, dx, dy, dz, dt, Re, Nx, Ny, Nz, Mz, ux_hat_d_shift, uy_hat_d_shift, uz_hat_d_shift,  ux_hat_d_1, uy_hat_d_1, uz_hat_d_1,  ux_hat_d_2, uy_hat_d_2, uz_hat_d_2,  ux_hat_d_3, uy_hat_d_3, uz_hat_d_3,  fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d,  kx_nabla_d,  ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d,  AM_12_d, AM_13_d, AM_23_d);
    }
    real rhs_x_shift, rhs_y_shift, rhs_z_shift;
    real x_0_shift, y_0_shift, z_0_shift;
    return_vector3_solution(j_fixed,  k_fixed, l_fixed, dimGrid, dimBlock, Nx, Ny, Nz, ux_hat_d_shift, uy_hat_d_shift, uz_hat_d_shift, ux_d_plane, uy_d_plane, uz_d_plane, ux, uy, uz, &x_0_shift, &y_0_shift, &z_0_shift);
    return_vector3_RHS(dimGrid,  dimBlock,  dimGrid_C, dimBlock_C, dx, dy, dz, Re, Nx, Ny,  Nz, Mz, ux_hat_d_shift, uy_hat_d_shift, uz_hat_d_shift,  fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d, kx_nabla_d,  ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d, AM_12_d, AM_13_d, AM_23_d, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, ux_d_plane, uy_d_plane, uz_d_plane, ux, uy, uz, j_fixed,  k_fixed, l_fixed, &rhs_x_shift, &rhs_y_shift, &rhs_z_shift);

    //copy_arrays(dimGrid_C, dimBlock_C, Nx, Ny, Nz, ux_hat_d_shift, uy_hat_d_shift, uz_hat_d_shift, ux_hat_d, uy_hat_d, uz_hat_d);
/*
    =============================================
    ENDS
    =============================================
*/

    real radius=0.01; //0.000001;
    int size=Nrad*Nphi;
 
    construct_plane_rectangular(Nrad, Nphi, x_loc, y_loc, z_loc, radius);


    rotate_plane(rhs_x, rhs_y, rhs_z, 0.0, 0.0, 1.0, Matrix, size, x_loc, y_loc, z_loc, p_x, p_y, p_z);
    translate_plane(x_0, y_0, z_0, size, p_x, p_y, p_z);


    real tvec_x=p_x[0]-p_x[Nphi*Nrad-3];
    real tvec_y=p_y[0]-p_y[Nphi*Nrad-3];
    real tvec_z=p_z[0]-p_z[Nphi*Nrad-3];
    
    real ivec_x=x_loc[0]-x_loc[Nphi*Nrad-3];
    real ivec_y=y_loc[0]-y_loc[Nphi*Nrad-3];
    real ivec_z=z_loc[0]-z_loc[Nphi*Nrad-3];

    printf("\n[%lf %lf %lf]->([%lf %lf %lf],[%lf %lf %lf]) plane test=%le \n", ivec_x, ivec_y, ivec_z, tvec_x, tvec_y, tvec_z, rhs_x, rhs_y, rhs_z, vector3_dot_product(rhs_x, rhs_y, rhs_z,tvec_x, tvec_y, tvec_z) );


    debug_plot_points("res_3D_0.dat", size, x_loc, y_loc, z_loc);
    debug_plot_points("res_3D.dat", size, p_x, p_y, p_z);
    plot_points_pos("res_3D.pos", size, p_x, p_y, p_z);
    debug_plot_vector("normal.dat", x_0, y_0, z_0, rhs_x, rhs_y, rhs_z, 1.0);
    

   
    real x_prev=0.0, y_prev=0.0, z_prev=0.0;
    real x_next=0.0, y_next=0.0, z_next=0.0;


    for (int j = 0; j < number_of_points; ++j){
        x_prev=p_x[j];
        y_prev=p_y[j];
        z_prev=p_z[j];


        construct_physical_vector(dimGrid, dimBlock, Nx, Ny, Nz, j_fixed, k_fixed, l_fixed,  x_prev, y_prev, z_prev, /* original solution */ ux_hat_d, uy_hat_d, uz_hat_d,/* ends */ ux_d_plane, uy_d_plane, uz_d_plane, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane);
       
        return_vector3_solution(j_fixed,  k_fixed, l_fixed, dimGrid, dimBlock, Nx, Ny, Nz, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, ux_d_plane, uy_d_plane, uz_d_plane, ux, uy, uz, &x_next, &y_next, &z_next);

        if( (std::fabs(x_prev-x_next)>1.0E-10)||(std::fabs(y_prev-y_next)>1.0E-10)||(std::fabs(z_prev-z_next)>1.0E-10) ){
            printf("\nWarning - non matching points at j=%i \n",j);
        }

        FILE *stream;
        char f1_name[100];
        sprintf(f1_name, "test_point_%i.dat",j); 
        stream=fopen(f1_name, "w" );
        bool stop_flag=false;
        int steps=0;
        fprintf( stream, "%.16le %.16le %.16le\n", x_prev, y_prev, z_prev); 
        int count_stop_flags=0;
        real x_0_subs=x_0_shift, y_0_subs=y_0_shift, z_0_subs=z_0_shift;
        real rhs_x_subs=rhs_x_shift, rhs_y_subs=rhs_y_shift, rhs_z_subs=rhs_z_shift;
        //real x_0_subs=x_0, y_0_subs=y_0, z_0_subs=z_0;
        //real rhs_x_subs=rhs_x, rhs_y_subs=rhs_y, rhs_z_subs=rhs_z;

        while(!stop_flag){
            
            stop_flag = find_intersection(steps, x_0_subs, y_0_subs, z_0_subs, &x_next, x_prev, &y_next, y_prev, &z_next, z_prev, rhs_x_subs, rhs_y_subs, rhs_z_subs, j_fixed, k_fixed, l_fixed, dimGrid, dimBlock, dimGrid_C, dimBlock_C, dx, dy, dz, dt, Re, Nx, Ny, Nz, Mz, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, ux_hat_d_plane_back, uy_hat_d_plane_back, uz_hat_d_plane_back, ux_hat_d_1, uy_hat_d_1, uz_hat_d_1,  ux_hat_d_2, uy_hat_d_2, uz_hat_d_2,  ux_hat_d_3, uy_hat_d_3, uz_hat_d_3,  fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d,  kx_nabla_d,  ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d,  AM_12_d, AM_13_d, AM_23_d, ux_d_plane, uy_d_plane, uz_d_plane, ux, uy, uz); 

            fprintf( stream, "%.16le %.16le %.16le\n", x_next, y_next, z_next); 
            x_prev=x_next;
            y_prev=y_next;
            z_prev=z_next;

/*            if(stop_flag){
                count_stop_flags++;
                if(count_stop_flags==1){
                    stop_flag=false;
                    p_x[j]=x_next;
                    p_y[j]=y_next;
                    p_z[j]=z_next;
                    real x_0_subs=x_0; y_0_subs=y_0; z_0_subs=z_0;
                    rhs_x_subs=rhs_x; rhs_y_subs=rhs_y; rhs_z_subs=rhs_z;
                }
            }
*/



            printf("[ %i ]   \r", steps);
            steps++;
            
        }
        fclose(stream);
        printf("\n");
        p_x[j]=x_next;
        p_y[j]=y_next;
        p_z[j]=z_next;

        //advance solution!!!
        copy_arrays(dimGrid_C, dimBlock_C, Nx, Ny, Nz, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, ux_hat_d, uy_hat_d, uz_hat_d);    
    }
    


    debug_plot_points("res_3D_1.dat", size, p_x, p_y, p_z);
    plot_points_pos("res_3D_1.pos", size, p_x, p_y, p_z);
    
    double test_vec_x0=0.0,test_vec_y0=0.0,test_vec_z0=1.0;
    double test_vec_x,test_vec_y,test_vec_z;
    rotate_plane(rhs_x, rhs_y, rhs_z, 0, 0, 1.0, Matrix, 1, &test_vec_x0, &test_vec_y0, &test_vec_z0, &test_vec_x, &test_vec_y, &test_vec_z);
    
    //translate_plane(x0, y0, z0, 1, &test_vec_x, &test_vec_y, &test_vec_z);
    debug_plot_vector("normal_1.dat", x_0, y_0, z_0, test_vec_x, test_vec_y, test_vec_z, 1.0);

    debug_plot_vectors("vectors.dat", size, p_x, p_y, p_z, v_x, v_y, v_z, 1.0);

    device_deallocate_all_complex(3, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane);
    device_deallocate_all_complex(3, ux_hat_d_plane_back, uy_hat_d_plane_back, uz_hat_d_plane_back);
    device_deallocate_all_complex(3, ux_hat_d_shift, uy_hat_d_shift, uz_hat_d_shift);
    device_deallocate_all_real(3, ux_d_plane, uy_d_plane, uz_d_plane);

    free(ux);
    free(uy);
    free(uz);

    free(x_loc);
    free(y_loc);
    free(p_x);
    free(p_y);
    free(p_z);
    free(Matrix);   
    free(vx_loc);
    free(vy_loc);
    free(v_x);
    free(v_y);
    free(v_z);  

}




bool find_intersection_curl(int steps, real x_0, real y_0, real z_0,  real *x_next, real x_prev, real *y_next, real y_prev, real *z_next, real z_prev, real rhs_x, real rhs_y, real rhs_z, int j_fixed, int k_fixed, int l_fixed, dim3 dimGrid, dim3 dimBlock, dim3 dimGrid_C, dim3 dimBlock_C, real dx, real dy, real dz, real dt, real Re, int Nx, int Ny, int Nz, int Mz, cudaComplex *ux_hat_d, cudaComplex *uy_hat_d, cudaComplex *uz_hat_d, cudaComplex *ux_hat_d_back, cudaComplex *uy_hat_d_back, cudaComplex *uz_hat_d_back, cudaComplex *ux_hat_d_1, cudaComplex *uy_hat_d_1, cudaComplex *uz_hat_d_1,  cudaComplex *ux_hat_d_2, cudaComplex *uy_hat_d_2, cudaComplex *uz_hat_d_2,  cudaComplex *ux_hat_d_3, cudaComplex *uy_hat_d_3, cudaComplex *uz_hat_d_3,  cudaComplex *fx_hat_d, cudaComplex *fy_hat_d, cudaComplex *fz_hat_d, cudaComplex *Qx_hat_d, cudaComplex *Qy_hat_d, cudaComplex *Qz_hat_d, cudaComplex *div_hat_d, real* kx_nabla_d, real* ky_nabla_d, real *kz_nabla_d, real *din_diffusion_d, real *din_poisson_d, real *AM_11_d, real *AM_22_d, real *AM_33_d,  real *AM_12_d, real *AM_13_d, real *AM_23_d, real *ux_d, real *uy_d, real *uz_d,  real *rot_x_d, real *rot_y_d, real *rot_z_d,  real *rot_x, real *rot_y, real *rot_z)
{

    const real rho=2.0e-1;
    bool return_flag=false;


    //note - "ux_hat_d_plane_back, uy_hat_d_plane_back, uz_hat_d_plane_back" are having previous timestep stored!

    copy_arrays(dimGrid_C, dimBlock_C, Nx, Ny, Nz,  ux_hat_d, uy_hat_d, uz_hat_d, ux_hat_d_back, uy_hat_d_back, uz_hat_d_back);

    single_forward_step_curl(dimGrid, dimBlock, dimGrid_C, dimBlock_C, dx, dy, dz, dt, Re, Nx, Ny, Nz, Mz, ux_hat_d, uy_hat_d, uz_hat_d, ux_hat_d_1, uy_hat_d_1, uz_hat_d_1,  ux_hat_d_2, uy_hat_d_2, uz_hat_d_2,  ux_hat_d_3, uy_hat_d_3, uz_hat_d_3,  fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d, kx_nabla_d, ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d,  AM_12_d, AM_13_d, AM_23_d, ux_d, uy_d, uz_d, rot_x_d, rot_y_d, rot_z_d,  rot_x, rot_y, rot_z, j_fixed, k_fixed, l_fixed, x_next, y_next, z_next);



    real test_vec_x=x_next[0]-x_prev, test_vec_y=y_next[0]-y_prev, test_vec_z=z_next[0]-z_prev;

    //crosses plane
    real sign_1=test_plane_location(rhs_x, rhs_y, rhs_z, x_0, y_0, z_0, x_prev, y_prev, z_prev);
    real sign_2=test_plane_location(rhs_x, rhs_y, rhs_z, x_0, y_0, z_0, x_next[0], y_next[0], z_next[0]); 
    
    //in the same direction
    real sign_3=vector3_dot_product(rhs_x, rhs_y, rhs_z, test_vec_x, test_vec_y, test_vec_z); 
    
    real vec_x=x_next[0]-x_0;
    real vec_y=y_next[0]-y_0;
    real vec_z=z_next[0]-z_0;
     //in the ball_rho
    real vec_norm=vector3_norm(vec_x, vec_y, vec_z);


    //if(vec_norm<rho){
    //    printf("||v||<rho %le \n",vec_norm);
    //}
    //if(sign_1*sign_2<0.0){
    //    printf("s1*s2<0 %le \n",sign_1*sign_2);
    //}
    //if(sign_3>0.0){
    //    printf("s3>0 %le \n",sign_3);
    //}


    if((steps>3)&&(vec_norm<rho)&&(sign_1*sign_2<0.0)&&(sign_3>0.0)){
        real dt1=dt;
        real xn1=x_prev, yn1=y_prev, zn1=z_prev;
        real err_s=test_plane_location(rhs_x, rhs_y, rhs_z, x_0, y_0, z_0, x_next[0], y_next[0], z_next[0]);
        real del_s=test_plane_location(rhs_x, rhs_y, rhs_z, x_0, y_0, z_0, x_prev, y_prev, z_prev);
        real err=std::fabs(err_s);
        real del=std::fabs(del_s);
        
        int iter=0;
        real a_val=0.0;
        real b_val=dt1;
        while((std::fabs(err)>1.0e-12)&&(iter<500)){                        
            iter++;
            
            real m_val=0.5*(b_val-a_val);
            //dt1=dt1*(del)/(del+err);
            

            //restore previous step!
            copy_arrays(dimGrid_C, dimBlock_C, Nx, Ny, Nz, ux_hat_d_back, uy_hat_d_back, uz_hat_d_back,  ux_hat_d, uy_hat_d, uz_hat_d);
            single_forward_step_curl(dimGrid, dimBlock, dimGrid_C, dimBlock_C,  dx, dy, dz, /*!*/m_val/*!*/, Re,  Nx,  Ny,  Nz,  Mz, ux_hat_d, uy_hat_d, uz_hat_d, ux_hat_d_1, uy_hat_d_1, uz_hat_d_1,  ux_hat_d_2, uy_hat_d_2, uz_hat_d_2,  ux_hat_d_3, uy_hat_d_3, uz_hat_d_3,  fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d, kx_nabla_d, ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d,  AM_12_d, AM_13_d, AM_23_d, ux_d, uy_d, uz_d, rot_x_d, rot_y_d, rot_z_d,  rot_x, rot_y, rot_z, j_fixed, k_fixed, l_fixed, &xn1, &yn1, &zn1);


            err=test_plane_location(rhs_x, rhs_y, rhs_z, x_0, y_0, z_0, xn1, yn1, zn1);

            if(err>0.0){
                b_val=m_val;
            }
            else{
                a_val=m_val;
                //shift base point
                copy_arrays(dimGrid_C, dimBlock_C, Nx, Ny, Nz,  ux_hat_d, uy_hat_d, uz_hat_d, ux_hat_d_back, uy_hat_d_back, uz_hat_d_back);      
            }

        }
        printf("\{%le,%i\}", err, iter);
        x_next[0]=xn1; y_next[0]=yn1; z_next[0]=zn1;
        return_flag=true;
    }

    return return_flag;
}


void execute_sections(int j_fixed, int k_fixed, int l_fixed, dim3 dimGrid, dim3 dimBlock, dim3 dimGrid_C, dim3 dimBlock_C, real dx, real dy, real dz, real dt, real Re, int Nx, int Ny, int Nz, int Mz, cudaComplex *ux_hat_d, cudaComplex *uy_hat_d, cudaComplex *uz_hat_d, cudaComplex *ux_hat_d_1, cudaComplex *uy_hat_d_1, cudaComplex *uz_hat_d_1,  cudaComplex *ux_hat_d_2, cudaComplex *uy_hat_d_2, cudaComplex *uz_hat_d_2,  cudaComplex *ux_hat_d_3, cudaComplex *uy_hat_d_3, cudaComplex *uz_hat_d_3,  cudaComplex *fx_hat_d, cudaComplex *fy_hat_d, cudaComplex *fz_hat_d, cudaComplex *Qx_hat_d, cudaComplex *Qy_hat_d, cudaComplex *Qz_hat_d, cudaComplex *div_hat_d, real* kx_nabla_d, real* ky_nabla_d, real *kz_nabla_d, real *din_diffusion_d, real *din_poisson_d, real *AM_11_d, real *AM_22_d, real *AM_33_d,  real *AM_12_d, real *AM_13_d, real *AM_23_d)
{

    real x_0, y_0, z_0;
    real rhs_x, rhs_y, rhs_z;

   
    real *ux, *uy, *uz;
    allocate_real(Nx, Ny, Nz, 3, &ux, &uy, &uz);

    cudaComplex *ux_hat_d_plane, *uy_hat_d_plane, *uz_hat_d_plane;
    cudaComplex *ux_hat_d_plane_back, *uy_hat_d_plane_back, *uz_hat_d_plane_back;
    cudaComplex *ux_hat_d_shift, *uy_hat_d_shift, *uz_hat_d_shift;
    real *ux_d_plane, *uy_d_plane, *uz_d_plane;
    //arrays for section storage
    real *ux_d_section, *uy_d_section, *uz_d_section;
    real *ux_section, *uy_section, *uz_section;
    real *rot_x_d, *rot_y_d, *rot_z_d;  
    real *rot_x, *rot_y, *rot_z;

    device_allocate_all_complex(Nx, Ny, Mz, 3, &ux_hat_d_plane, &uy_hat_d_plane, &uz_hat_d_plane);
    device_allocate_all_complex(Nx, Ny, Mz, 3, &ux_hat_d_plane_back, &uy_hat_d_plane_back, &uz_hat_d_plane_back);
    device_allocate_all_complex(Nx, Ny, Mz, 3, &ux_hat_d_shift, &uy_hat_d_shift, &uz_hat_d_shift);
    device_allocate_all_real(Nx, Ny, Nz, 3, &ux_d_plane, &uy_d_plane, &uz_d_plane);
    device_allocate_all_real(Nx, Ny, Nz, 3, &rot_x_d, &rot_y_d, &rot_z_d);
    allocate_real(Nx, Ny, Nz, 3, &rot_x, &rot_y, &rot_z);


    //obtaining the RHS vector at a currect solution point
    return_vector3_RHS_curl(dimGrid,  dimBlock,  dimGrid_C, dimBlock_C, dx, dy, dz, Re, Nx, Ny,  Nz, Mz, ux_hat_d, uy_hat_d, uz_hat_d,  fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d, kx_nabla_d,  ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d, AM_12_d, AM_13_d, AM_23_d, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, ux_d_plane, uy_d_plane, uz_d_plane, rot_x_d, rot_y_d, rot_z_d, rot_x, rot_y, rot_z, j_fixed,  k_fixed, l_fixed, &rhs_x, &rhs_y, &rhs_z);

    //get a point of x0,y0,z0 from the solution
    return_vector3_solution_curl(j_fixed,  k_fixed, l_fixed, dimGrid, dimBlock, Nx, Ny, Nz, dx, dy, dz, ux_hat_d, uy_hat_d, uz_hat_d, ux_d_plane, uy_d_plane, uz_d_plane, rot_x_d, rot_y_d, rot_z_d, rot_x, rot_y, rot_z, &x_0, &y_0, &z_0);

    debug_plot_vector("normal.dat", x_0, y_0, z_0, rhs_x, rhs_y, rhs_z, 1.0);
    
    real x_prev=x_0, y_prev=y_0, z_prev=z_0;
    real x_next=0.0, y_next=0.0, z_next=0.0;

    int number_of_intersections=100;

    device_allocate_all_real(Nx, Ny, Nz, 3, &ux_d_section, &uy_d_section, &uz_d_section);
    allocate_real(Nx, Ny, Nz*number_of_intersections, 3, &ux_section, &uy_section, &uz_section);



    FILE *stream;
    char f1_name[100];
    sprintf(f1_name, "test_point.dat");        
    stream=fopen(f1_name, "w" );
    for (int t = 0; t < number_of_intersections; ++t){

        copy_arrays(dimGrid_C, dimBlock_C, Nx, Ny, Nz, ux_hat_d, uy_hat_d, uz_hat_d, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane);
        copy_arrays(dimGrid_C, dimBlock_C, Nx, Ny, Nz, ux_hat_d, uy_hat_d, uz_hat_d, ux_hat_d_plane_back, uy_hat_d_plane_back, uz_hat_d_plane_back);  
  

        bool stop_flag=false;
        int steps=0;

        while(!stop_flag){
            
            stop_flag = find_intersection_curl(steps, x_0, y_0, z_0, &x_next, x_prev, &y_next, y_prev, &z_next, z_prev, rhs_x, rhs_y, rhs_z, j_fixed, k_fixed, l_fixed, dimGrid, dimBlock, dimGrid_C, dimBlock_C, dx, dy, dz, dt, Re, Nx, Ny, Nz, Mz, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, ux_hat_d_plane_back, uy_hat_d_plane_back, uz_hat_d_plane_back, ux_hat_d_1, uy_hat_d_1, uz_hat_d_1,  ux_hat_d_2, uy_hat_d_2, uz_hat_d_2,  ux_hat_d_3, uy_hat_d_3, uz_hat_d_3,  fx_hat_d, fy_hat_d, fz_hat_d, Qx_hat_d, Qy_hat_d, Qz_hat_d, div_hat_d,  kx_nabla_d,  ky_nabla_d, kz_nabla_d, din_diffusion_d, din_poisson_d, AM_11_d, AM_22_d, AM_33_d,  AM_12_d, AM_13_d, AM_23_d, ux_d_plane, uy_d_plane, uz_d_plane, rot_x_d, rot_y_d, rot_z_d,  rot_x, rot_y, rot_z);

            x_prev=x_next;
            y_prev=y_next;
            z_prev=z_next;

            printf("[ %i ]   \r", steps);
            steps++;

        }

        velocity_to_double(dimGrid, dimBlock, Nx, Ny, Nz, ux_hat_d_plane, ux_d_plane, uy_hat_d_plane, uy_d_plane, uz_hat_d_plane, uz_d_plane);
        host_device_real_cpy(ux, ux_d_plane, Nx, Ny, Nz);
        host_device_real_cpy(uy, uy_d_plane, Nx, Ny, Nz);
        host_device_real_cpy(uz, uz_d_plane, Nx, Ny, Nz);

        //fprintf( stream, "%.16le %.16le %.16le\n", x_prev, y_prev, z_prev);
        for(int j=0;j<Nx;j++){
            for(int k=0;k<Ny;k++){
                for(int l=0;l<Nz;l++){
                    fprintf( stream, "%.16le %.16le %.16le ", ux[IN(j,k,l)], uy[IN(j,k,l)], uz[IN(j,k,l)]);
                }
            }
        }
        fprintf( stream, "\n");
        printf("\n");        
        //advance solution!!!
        copy_arrays(dimGrid_C, dimBlock_C, Nx, Ny, Nz, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane, ux_hat_d, uy_hat_d, uz_hat_d);    
        
    }
    
    fclose(stream);

/*    
    FILE *stream;
    char f1_name[100];
    sprintf(f1_name, "test_point_%i.dat",j); 
    stream=fopen(f1_name, "w" );
    fprintf( stream, "%.16le %.16le %.16le\n", x_prev, y_prev, z_prev); 
    fclose(stream);
*/

   

    device_deallocate_all_complex(3, ux_hat_d_plane, uy_hat_d_plane, uz_hat_d_plane);
    device_deallocate_all_complex(3, ux_hat_d_plane_back, uy_hat_d_plane_back, uz_hat_d_plane_back);
    device_deallocate_all_complex(3, ux_hat_d_shift, uy_hat_d_shift, uz_hat_d_shift);
    device_deallocate_all_real(3, ux_d_plane, uy_d_plane, uz_d_plane);
    device_deallocate_all_real(3, ux_d_section, uy_d_section, uz_d_section);
    device_deallocate_all_real(3, rot_x_d, rot_y_d, rot_z_d);

    free(ux);
    free(uy);
    free(uz);

    free(ux_section);
    free(uy_section);
    free(uz_section);

    free(rot_x);
    free(rot_y);
    free(rot_z);

}

